/*
 *  Cervnet is a c++ library which implements Kronecker recurrent units (KRU) and several other recurrent neural networks.  
 *
 *  Copyright (c) 2016 Idiap Research Institute, http://www.idiap.ch/
 *  Written by Cijo Jose <cijo.jose@alumni.epfl.ch>
 *
 *  This file is part of Cervnet.
 *
 *  Cervnet is free software: you can redistribute it and/or modify it
 *  under the terms of the GNU General Public License version 3 as
 *  published by the Free Software Foundation.
 *
 *  Cervnet is distributed in the hope that it will be useful, but WITHOUT
 *  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 *  or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public
 *  License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with selector.  If not, see <http://www.gnu.org/licenses/>.
 *
 */
// nvcc KroneckerForwardBackward.cu -ccbin=g++-4.9  -std=c++11 -lcublas -o KroneckerForwardBackward.bin

#include <iostream>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <cassert>
#include <cstring>
#include <algorithm>

#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hip/driver_types.h>
#include <hipblas.h>
const uint32_t CUDA_NUM_THREADS = 1024;
inline uint32_t CUDA_GET_BLOCKS(const uint32_t N) {
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}
#define CUDA_CHECK(condition) assert(condition == hipSuccess)


void cpu_kronecker_forward_kernel(int M, int N, int rowk,
                                       int colk, int stride,
                                       const float* W_k, const float* X, float * Y) {
  int index = 0;
  for(int m = 0; m < M; m++){
    const float* X_m = X + m * N;
    for(int p = 0; p < rowk; p++){
      for(int q = 0; q < stride; q++){
        Y[index] = 0;
        for(int r= 0; r < colk; r++){
          Y[index] += X_m[r * stride + q] * W_k[p * colk + r];
        }
        index++;
      }
    }
  }
}

/*

The code computes Y = XW^T

Input matrix X \in R^{M \times N}

Wsize is an array of size 2K containing the number of rows and columns each
Kronecker factors {P_0, Q_0 ..., P_{K−1}, Q_{K-1}} : \prod_{k=0}^{K−1} P_k = D : : \prod_{k=0}^{K−1} Q_k = N
                 
Kronecker factors \{W_0, ..., W_{K−1}\} : W_k \in R^{P_k\times Q_k},

Ysize is an array of length K such that \sum_{k = 0}^{K - 1} Ysize[i] = size of Y, 1.e each entry gives the memory required to
store the product with corresponding Kronecker factors.  

size of Y =  O(MDK) = \sum_{k = 0}^{K - 1} Ysize[i]

*/

void cpu_kronecker_forward(int M, int N, const float *X,
                                int K, const int* Wsize,
                                const float* W, const int* Ysize, float *Y) {
  int offset = 0, k;
  const float* X_k = X;
  float *Y_k = Y;
  for(k = 0; k <  K; k++)  {
    int rowk = Wsize[2 * k];
    int colk = Wsize[2 * k + 1];
    int stride = N / colk;
    const float* W_k = W + offset;
    if(k > 0) {
      assert(Ysize[k-1] == M * N);
    }
    cpu_kronecker_forward_kernel(M, N, rowk,
                                                colk, stride, W_k,
                                                X_k, Y_k);
    N = stride;
    M = M * rowk;
    offset += rowk * colk;
    X_k = Y_k;
    assert(Ysize[k] == M * N);
    Y_k += Ysize[k];
  }
}


__global__ void gpu_kronecker_forward_kernel(int M, int N, int rowk,
                                                  int colk, int stride,
                                                  const float* W_k, const float* X, float * Y) {
  int TH = threadIdx.x + blockIdx.x * blockDim.x;
  if(TH < M * rowk * stride) {
    int m = TH / (rowk * stride);
    int pq = TH % (rowk * stride);
    int p = pq / stride;
    int q = pq % stride;
    const float* X_m = X + m * N;
    Y[TH] = 0;
    for(int r= 0; r < colk; r++){
      Y[TH] += X_m[r * stride + q] * W_k[p * colk + r];
    }
  }
}

/*
  size of Y is O(MDK)
*/
void gpu_kronecker_forward(int M, int N, const float *X,
                                int K, const int* Wsize,
                                const float* W, const int* Ysize, float *Y) {
  int offset = 0, k;
  const float* X_k = X;
  float *Y_k = Y;
  for(k = 0; k <  K; k++)  {
    int rowk = Wsize[2 * k];
    int colk = Wsize[2 * k + 1];
    int stride = N / colk;
    const float* W_k = W + offset;
    if(k > 0) {
      assert(Ysize[k-1] == M * N);
    }
    gpu_kronecker_forward_kernel<<<CUDA_GET_BLOCKS(M * rowk * stride), CUDA_NUM_THREADS>>>(M, N, rowk,
                                                                                                colk, stride, W_k,
                                                                                                X_k, Y_k);
    N = stride;
    M = M * rowk;
    offset += rowk * colk;
    X_k = Y_k;
    assert(Ysize[k] == M * N);
    Y_k += Ysize[k];
  }
}


void cpu_kronecker_backward_kernel1(int M, int N, int rowk, int colk,
                                                   int stride, const float* X, const float* gradY, float* gradW_k) {
  int index = 0;
  for(int m = 0; m < M; m++){
    const float* X_m = X + m * N;
    for(int p = 0; p < rowk; p++){
      for(int q = 0; q < stride; q++){
        for(int r= 0; r < colk; r++){
          gradW_k[p * colk + r] += X_m[r * stride + q] * gradY[index];
        }
        index++;
      }
    }
  }
}

void cpu_kronecker_backward_kernel2(int M, int N, int rowk,
                                         int colk, int stride,
                                         const float* W_k, const float* gradY, float * gradX) {
  int index = 0;
  for(int m = 0; m < M; m++){
    const float* gradY_m = gradY + m * N;
    for(int r= 0; r < colk; r++){
      for(int q = 0; q < stride; q++){
        gradX[index] = 0;
        for(int p = 0; p < rowk; p++){
          gradX[index] += gradY_m[p * stride + q] * W_k[p * colk + r];
        }
        index++;
      }
    }
  }
  //cout<<index<<endl;
}

/*
Y = XW^T = forward
gradX = gradY * W
gradW += gradY^T * X
size of Y is O(MDK)
I use the memory of Y to do intermediate computations so the
Y values are  not preserved
*/
void cpu_kronecker_backward(int M, int N, int D, const float *X,
                                 int K, const int* Worder,
                                 const float* W, const float* gradY,
                                 const int* Ysize, float* Y, float* gradW, float* gradX) {
  int P  = M * D;
  int stride = 1;
  int stride1 = 1;
  int offsetW = 0;
  int offsetY = 0;
  for(int k = 0 ; k < K; k++) {
    offsetW += Worder[2* k] * Worder[2 * k + 1];
    offsetY += Ysize[k];
  }
  const float* gradY_k = gradY;
  for(int k = K - 1; k >= 0; k--) {
    offsetY -= Ysize[k];
    int rowk = Worder[2 * k];
    int colk = Worder[2 * k + 1];
    int Q = stride * colk;
    int S = stride1 * rowk;
    P /= rowk;
    offsetW -= rowk * colk;
    const float* W_k = W + offsetW;
    float* gradW_k = gradW + offsetW;
    const float* X_k = nullptr;
    float* gradX_k = nullptr;
    if(k == 0) {
      X_k = X;
      gradX_k = gradX;

    }
    else {
      gradX_k =  Y + offsetY - Ysize[k - 1];
      X_k = gradX_k;
    }
    cpu_kronecker_backward_kernel1(P, Q, rowk, colk, stride, X_k,
                                        gradY_k, gradW_k);
    cpu_kronecker_backward_kernel2(P, S, rowk, colk, stride, W_k,
                                        gradY_k, gradX_k);
    gradY_k = gradX_k;
    stride = Q;
    stride1 = S * colk / rowk;
  }
}

__global__ void gpu_kronecker_backward_kernel1(int M, int N, int rowk, int colk,
                                                    int stride, const float* X, const float* gradY, float* gradW_k) {
  int TH = threadIdx.x + blockIdx.x * blockDim.x;
  if(TH < M * rowk * stride) {
    int m = TH / (rowk * stride);
    int pq = TH % (rowk * stride);
    int p = pq / stride;
    int q = pq % stride;
    const float* X_m = X + m * N;
    for(int r= 0; r < colk; r++){
      atomicAdd(&gradW_k[p * colk + r], X_m[r * stride + q] * gradY[TH]);
    }
  }
}

__global__  void gpu_kronecker_backward_kernel2(int M, int N, int rowk,
                                                     int colk, int stride,
                                                     const float* W_k, const float* gradY, float * gradX) {
  int TH = threadIdx.x + blockIdx.x * blockDim.x;
  if(TH < M * colk * stride) {
    int m = TH / (colk * stride);
    int rq = TH % (colk * stride);
    int r = rq / stride;
    int q = rq % stride;
    const float* gradY_m = gradY + m * N;
    gradX[TH] = 0;
    for(int p = 0; p < rowk; p++){
      gradX[TH] += gradY_m[p * stride + q] * W_k[p * colk + r];
    }
  }
}

/*
Y = XW = forward
gradX = gradY * W^T
gradW += X^T * gradY
size of Y is O(MDK)
I use the memory of Y to do intermediate computations so the
Y values are  not preserved
*/
void gpu_kronecker_backward(int M, int N, int D, const float *X,
                                 int K, const int* Worder,
                                 const float* W, const float* gradY,
                                 const int* Ysize, float* Y, float* gradW, float* gradX) {
  int P  = M * D;
  int stride = 1;
  int stride1 = 1;
  int offsetW = 0;
  int offsetY = 0;
  for(int k = 0 ; k < K; k++) {
    offsetW += Worder[2* k] * Worder[2 * k + 1];
    offsetY += Ysize[k];
  }
  const float* gradY_k = gradY;
  for(int k = K - 1; k >= 0; k--) {
    offsetY -= Ysize[k];
    int rowk = Worder[2 * k];
    int colk = Worder[2 * k + 1];
    int Q = stride * colk;
    int S = stride1 * rowk;
    P /= rowk;
    offsetW -= rowk * colk;
    const float* W_k = W + offsetW;
    float* gradW_k = gradW + offsetW;
    const float* X_k = nullptr;
    float* gradX_k = nullptr;
    if(k == 0) {
      X_k = X;
      gradX_k = gradX;
    }
    else {
      gradX_k =  Y + offsetY - Ysize[k - 1];
      X_k = gradX_k;
    }
    gpu_kronecker_backward_kernel1<<<CUDA_GET_BLOCKS(P * rowk * stride),
        CUDA_NUM_THREADS>>>(P, Q, rowk, colk, stride, X_k, gradY_k, gradW_k);
    gpu_kronecker_backward_kernel2<<<CUDA_GET_BLOCKS(P * colk * stride),
        CUDA_NUM_THREADS>>>(P, S, rowk, colk, stride, W_k, gradY_k, gradX_k);
    gradY_k = gradX_k;
    stride = Q;
    stride1 = S * colk / rowk;
  }
}


void sort(int N, int* arr) {
  int i, key, j;
  for (i = 1; i < N; i++) {
    key = arr[i];
    j = i-1;
    while (j >= 0 && arr[j] > key) {
      arr[j+1] = arr[j];
      j = j-1;
    }
    arr[j+1] = key;
  }
}

void sieve(int N, int** primes, int &num_primes) {
  assert(N >= 2);
  bool *A = new bool [N + 1];
  memset(A, 0, (N + 1)  * sizeof(bool));
  for(int i = 2; i <= sqrt(N); i++) {
    if(A[i] == false) {
      for(int  j = i * i; j <= N; j += i) {
        A[j] =  true;
      }
    }
  }
  num_primes = 0;
  for(int i = 2; i <= N; i++) {
    if(!A[i]) { num_primes++; }
  }
  (*primes) = new int [num_primes];
  int count  = 0;
  for(int i = 2; i <= N; i++) {
    if(!A[i]) {
      (*primes)[count] = i;
      count++;
    }
  }
  delete [] A;
}


/*
  Given a whole number N with >= M factors, the function returns M numbers
   such that the prduct of these M numbers is N.
   Moreover \sum_i^{M} factors_sizes[i]^2 is minimum.
*/
void get_factor_sizes(int N, int M,
                      int* factor_sizes) {
  std::vector<int> factors;
  int num_primes = 0;
  int* primes = nullptr;
  sieve(N, &primes, num_primes);
  for(int i =0; i < num_primes; i++) {
    int n = N;
    while (true) {
      if(n % primes[i] == 0 && n > 0) {
        factors.push_back(primes[i]);
        n /= primes[i];
      }
      else {
        break;
      }
    }
  }
  delete [] primes;
  assert(factors.size() >= M);
  if(factors.size() == M) {
    std::sort(factors.begin(), factors.end());
    for(int  i = 0; i < M; i++) {
      factor_sizes[i] = factors[i];
    }
  }
  else {
    int K = factors.size();
    int i =  0;
    while(K > M) {
      factors[i + 1] *= factors[i];
      std::sort(factors.begin(), factors.end());
      i++;
      K--;
    }
    for(int j = i; j < factors.size(); j++) {
      factor_sizes[j - i] = factors[j];
    }
  }
}



void knuth_shuffle(std::vector<int>& A) {
  for(int i = 0 ; i < A.size(); i++) {
    int r = i +  int((A.size() - i - 1) * (float(rand()) / float(RAND_MAX)));
    int tmp = A[i];
    A[i] = A[r];
    A[r] =  tmp;
  }
}


int kronecker_output_memory(int M, int N, int K, 
                            const int* Wsize, int* Ysize) {
  int size = 0;
  for(int  k =0; k < K; k++) {
    int rowk = Wsize[2 * k];
    int colk = Wsize[2 * k +  1];
    int stride = N / colk;
    N = stride;
    M = M * rowk;
    Ysize[k] = M * N;
    size += Ysize[k];
  }
  return size;
}



void kronecker_factor_sizes(int input_dim, int output_dim, int max_factors,
                            std::vector<int>& input_dim_factor, std::vector<int>& output_dim_factor) {

  int *input_dim_factors = new int [max_factors];
  int *output_dim_factors = new int [max_factors];
  get_factor_sizes(input_dim, max_factors, input_dim_factors);
  get_factor_sizes(output_dim, max_factors, output_dim_factors);
  for(int i = 0; i < max_factors; i++) {
    input_dim_factor.push_back(input_dim_factors[i]);
    output_dim_factor.push_back(output_dim_factors[i]);
  }
  delete [] input_dim_factors;
  delete [] output_dim_factors;

}

void print(int N , const float *X) {
  for(int i = 0; i < N; i++){
    std::cout<<X[i]<<" ";
  }
  std::cout<<std::endl<<std::endl;
}


float relative_error(int N,  const float* grad_analytic,  const float* grad_numerical) {
  float rel_error = 0;
  for(int i = 0 ; i < N; i++) {
    float a =  grad_analytic[i];
    float b =  grad_numerical[i];
    rel_error += fabs(a - b) / (fabs(a) + fabs(b));
  }
  rel_error /= float(N);
  return rel_error;
}


void forward_backward_checker(int M, int N, int D, int max_factors) {
  std::vector<int> factorsN, factorsD;
  kronecker_factor_sizes(N, D, max_factors, factorsN, factorsD);
  assert(factorsN.size() == factorsD.size());
  int numF = factorsN.size();
  int *Wsize = new int [numF * 2];
  int wsize = 0;
  //std::cout<<numF<<std::endl;
  for(int f =0; f < numF; f++) {
    Wsize[2 * f] = factorsD[f];
    Wsize[2 * f + 1] = factorsN[f];
    wsize += factorsD[f] * factorsN[f];
  }
  int* Ysize = new int [numF];
  int ysize = kronecker_output_memory(M, N, numF, Wsize, Ysize);

  hipblasHandle_t h;
  hipblasCreate(&h);
  
  float *W  =nullptr;
  CUDA_CHECK(hipHostMalloc(&W, wsize * sizeof(float), hipHostMallocDefault));
  float *X = nullptr;
  CUDA_CHECK(hipHostMalloc(&X, M * N * sizeof(float), hipHostMallocDefault));
  float *Y1 = new float [ysize];
  float *Y2 = new float [ysize];

  float *W_gpu  =nullptr;
  CUDA_CHECK(hipMalloc(&W_gpu, wsize * sizeof(float)));
  float *X_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&X_gpu, M * N * sizeof(float)));
  float *Y_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&Y_gpu, ysize * sizeof(float)));

  float *gradW_analytical  = new float [wsize];
  float *gradW_numerical  = new float [wsize];
  float *gradX_analytical = new float [M * N];
  float *gradX_numerical  = new float [M * N];

  float *Ypdx = new float [ysize];
  float *Ymdx = new float [ysize];
  float *gradY = new float [M * D]; 

  float *gradY_gpu = nullptr; 
  CUDA_CHECK(hipMalloc(&gradY_gpu, M * D * sizeof(float)));
  float *gradW_analytical_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&gradW_analytical_gpu, wsize * sizeof(float)));
  float *gradX_analytical_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&gradX_analytical_gpu, M * N * sizeof(float)));
  float *gradW_numerical_gpu = nullptr;
  CUDA_CHECK(hipHostMalloc(&gradW_numerical_gpu, wsize * sizeof(float), hipHostMallocDefault));
  float *gradX_numerical_gpu = nullptr;
  CUDA_CHECK(hipHostMalloc(&gradX_numerical_gpu, M * N * sizeof(float), hipHostMallocDefault));
  float *Ypdx_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&Ypdx_gpu, ysize * sizeof(float)));
  float *Ymdx_gpu = nullptr;
  CUDA_CHECK(hipMalloc(&Ymdx_gpu, ysize * sizeof(float)));

  for(int  i = 0; i < M * N; i++) {
    X[i] = float(rand()) / float(RAND_MAX);
  }
  for(int  i = 0; i < M * D; i++) {
    gradY[i] = float(rand()) / float(RAND_MAX);
  }
  for(int  i = 0; i < wsize; i++) {
    W[i] = float(rand()) / float(RAND_MAX);
  }
  CUDA_CHECK(hipMemcpy(W_gpu, W, wsize * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(X_gpu, X, M *  N * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gradY_gpu, gradY, M *  D * sizeof(float),
                        hipMemcpyHostToDevice));
  
  cpu_kronecker_forward(M, N, X, numF, Wsize,
                             W, Ysize, Y1);
  gpu_kronecker_forward(M, N, X_gpu, numF, Wsize,
                             W_gpu, Ysize, Y_gpu);
  CUDA_CHECK(hipMemcpy(Y2, Y_gpu, ysize * sizeof(float),
                        hipMemcpyDeviceToHost));
  std::cout<<"Relative error of forward pass cpu vs gpu "<<relative_error(ysize, Y1, Y2)<<std::endl;
  cpu_kronecker_forward(M, N, X, numF, Wsize,
                             W, Ysize, Y1);
  memset(gradW_analytical, 0, sizeof(float) * wsize);
  CUDA_CHECK(hipMemcpy(gradW_analytical_gpu, gradW_analytical, wsize * sizeof(float),
                        hipMemcpyHostToDevice));
  cpu_kronecker_backward(M, N, D, X, numF, Wsize, W,
                              gradY, Ysize, Y1, gradW_analytical, gradX_analytical);
  gpu_kronecker_backward(M, N, D, X, numF, Wsize, W,
                              gradY_gpu, Ysize, Y_gpu, gradW_analytical_gpu, gradX_analytical_gpu);

  CUDA_CHECK(hipMemcpy(gradW_numerical, gradW_analytical_gpu, wsize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(gradX_numerical, gradX_analytical_gpu, M * N * sizeof(float),
                        hipMemcpyDeviceToHost));
  std::cout<<"Relative error of backward pass on gradW cpu vs gpu "<<relative_error(wsize, gradW_numerical, gradW_analytical)<<std::endl;
  std::cout<<"Relative error of backward pass on gradX cpu vs gpu "<<relative_error(M * N, gradX_numerical, gradX_analytical)<<std::endl;


  
  /*
    Numerical gradient check follows.
  */
  int offset = ysize - M * D;
  float dx = 1e-2;
  for(int i =0 ; i < M * N; i++) {
    X[i] += dx;
    CUDA_CHECK(hipMemcpy(X_gpu + i, X + i, sizeof(float),
                          hipMemcpyHostToDevice));
    cpu_kronecker_forward(M, N, X, numF, Wsize, W, Ysize, Ypdx);
    gpu_kronecker_forward(M, N, X_gpu, numF, Wsize, W_gpu, Ysize, Ypdx_gpu);
    X[i] -= float(2) * dx;
    CUDA_CHECK(hipMemcpy(X_gpu + i, X + i, sizeof(float),
                          hipMemcpyHostToDevice));
    cpu_kronecker_forward(M, N, X, numF, Wsize, W, Ysize, Ymdx);
    gpu_kronecker_forward(M, N, X_gpu, numF, Wsize, W_gpu, Ysize, Ymdx_gpu);
    X[i] += dx;
    CUDA_CHECK(hipMemcpy(X_gpu + i, X + i, sizeof(float),
                          hipMemcpyHostToDevice));
    gradX_numerical[i] = 0;
    for(int  j  = 0; j < M * D; j++) {
      gradX_numerical[i] += (Ypdx[offset + j] - Ymdx[offset + j]) * gradY[j] / (float(2) * dx);
    }
    float sout;
    hipblasSdot(h, M * D, Ypdx_gpu + offset, 1,
               gradY_gpu, 1, &sout);
    gradX_numerical_gpu[i] = sout;
    hipblasSdot(h, M * D, Ymdx_gpu + offset, 1, gradY_gpu, 1, &sout);
    gradX_numerical_gpu[i] -= sout;
    gradX_numerical_gpu[i] /= (float(2) * dx);

  }

  for(int i =0 ; i < wsize; i++) {
    W[i] += dx;
    CUDA_CHECK(hipMemcpy(W_gpu + i, W + i, sizeof(float),
                          hipMemcpyHostToDevice));
    cpu_kronecker_forward(M, N, X, numF, Wsize, W, Ysize, Ypdx);
    gpu_kronecker_forward(M, N, X_gpu, numF, Wsize, W_gpu, Ysize, Ypdx_gpu);
    W[i] -= float(2) * dx;
    CUDA_CHECK(hipMemcpy(W_gpu + i, W + i, sizeof(float),
                          hipMemcpyHostToDevice));
    cpu_kronecker_forward(M, N, X, numF, Wsize, W, Ysize, Ymdx);
    gpu_kronecker_forward(M, N, X_gpu, numF, Wsize, W_gpu, Ysize, Ymdx_gpu);
    W[i] += dx;
    CUDA_CHECK(hipMemcpy(W_gpu + i, W + i, sizeof(float),
                          hipMemcpyHostToDevice));
    gradW_numerical[i] = 0;
    for(int  j  = 0; j < M * D; j++) {
      gradW_numerical[i] += (Ypdx[offset + j] - Ymdx[offset + j]) * gradY[j] / (float(2) * dx);
    }

    float sout;
    hipblasSdot(h, M * D, Ypdx_gpu + offset, 1,
               gradY_gpu, 1, &sout);
    gradW_numerical_gpu[i] = sout;
    hipblasSdot(h, M * D, Ymdx_gpu + offset, 1, gradY_gpu, 1, &sout);
    gradW_numerical_gpu[i] -= sout;
    gradW_numerical_gpu[i] /= (float(2) * dx);
  }

  std::cout<<"Relative error cpu gradW "<<relative_error(wsize, gradW_numerical, gradW_analytical)<<std::endl;
  std::cout<<"Relative error cpu gradX "<<relative_error(M * N, gradX_numerical, gradX_analytical)<<std::endl;

  CUDA_CHECK(hipMemcpy(gradW_analytical, gradW_analytical_gpu, wsize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(gradX_analytical, gradX_analytical_gpu, M * N * sizeof(float),
                        hipMemcpyDeviceToHost));

  std::cout<<"Relative error gpu gradW "<<relative_error(wsize, gradW_numerical_gpu, gradW_analytical)<<std::endl;
  std::cout<<"Relative error gpu gradX "<<relative_error(M * N, gradX_numerical_gpu, gradX_analytical)<<std::endl;


  //print(M * N, gradX_numerical);
  //print(M * N, gradX_analytical);

  delete [] Y1;
  delete [] Y2;
  delete [] Wsize;
  delete [] Ysize;
  delete [] gradW_analytical;
  delete [] gradW_numerical;
  delete [] gradX_analytical;
  delete [] gradX_numerical;
  delete [] Ypdx;
  delete [] Ymdx;
  delete [] gradY;
  CUDA_CHECK(hipHostFree(W));
  CUDA_CHECK(hipHostFree(X));
  CUDA_CHECK(hipFree(W_gpu));
  CUDA_CHECK(hipFree(X_gpu));
  CUDA_CHECK(hipFree(Y_gpu));

  CUDA_CHECK(hipFree(gradW_analytical_gpu));
  CUDA_CHECK(hipFree(gradX_analytical_gpu));
  CUDA_CHECK(hipHostFree(gradW_numerical_gpu));
  CUDA_CHECK(hipHostFree(gradX_numerical_gpu));
  CUDA_CHECK(hipFree(Ypdx_gpu));
  CUDA_CHECK(hipFree(Ymdx_gpu));  
}

int main() {

  forward_backward_checker(1, 16, 16, 3);
  forward_backward_checker(5, 32, 32, 4);
  forward_backward_checker(5, 32, 16, 4);
  forward_backward_checker(5, 16, 32, 4);
  forward_backward_checker(10, 32, 16, 1);
  forward_backward_checker(15, 16, 32, 3);
  forward_backward_checker(5, 16, 32, 1);
  forward_backward_checker(5, 64, 32, 1);
  forward_backward_checker(5, 64, 32, 5);
  /*
  forward_backward_checker(3, 128, 256, 5);
  forward_backward_checker(400, 4, 4, 2);
  forward_backward_checker(700, 200, 10000, 5);
  */
}
